#include "hip/hip_runtime.h"
#include "common/book.h"
#define N 10
/*void add(int *a, int *b, int *c) {
	int tid = 0; //To jest CPU nr zero, a wi�c zaczynamy od zera
	while (tid < N) {
		c[tid] = a[tid] + b[tid];
		tid += 1; // Mamy tylko jeden CPU, a wi�c zwi�kszamy o jeden
	}
}
int main(void) {
	int a[N], b[N], c[N];
	//Zape�nienie tablic a i b danymi za pomoc� CPU
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	add(a, b, c);
	// Wy�wietlenie wynik�w
	for (int i = 0; i < N; i++) {
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	return 0;*/

__global__ void add(int* a, int* b, int* c) {
	int tid = blockIdx.x; // Dzia�anie na danych znajduj�cych si� pod tym indeksem
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(void) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	// Alokacja pami�ci na GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
	//Zape�nienie tablic a i b na CPU
	
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}
	//Kopiowanie tablic a i b do GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int),
		hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int),
		hipMemcpyHostToDevice));
	add<<< N, 1 >>>(dev_a, dev_b, dev_c);
		// Kopiowanie tablicy c z GPU do CPU
		HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int),
			hipMemcpyDeviceToHost));
		// Wy�wietlenie wyniku
		for (int i = 0; i < N; i++) {
			printf("%d + %d = %d\n", a[i], b[i], c[i]);
		}
		// Zwolnienie pami�ci alokowanej na GPU
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_c);

	return 0;
}